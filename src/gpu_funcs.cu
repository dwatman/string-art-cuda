#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <stdint.h>
//#include <float.h>

// CUDA Runtime
#include <hip/hip_runtime.h>

#include "gpu_funcs.h"
#include "gpu_util.h"
#include "settings.h"

// GPU buffers
extern gpuData_t gpuData;

// Initialise gpuImgData_t structure and allocate buffers
int GpuInitBuffers(gpuData_t *gpuData, int widthIn, int heightIn) {
	printf("GpuInitBuffers\n");

	// Clear the structure in case of errors part way through initialisation
	memset(gpuData, 0, sizeof(gpuData_t));

	gpuData->srcWidth  = widthIn;
	gpuData->srcHeight = heightIn;
	gpuData->dstSize = DATA_SIZE;

	// Create stream for processing
	CUDA_CHECK(hipStreamCreate(&gpuData->stream));

	// Buffer for line data
	CUDA_CHECK(hipMalloc(&gpuData->lineData, NUM_LINES*4*sizeof(float)));

	// Buffers for image difference calculation
	CUDA_CHECK(hipMalloc((void**)&gpuData->partialSums, SUM_BLOCK_SIZE*SUM_BLOCK_SIZE*sizeof(double)));
	CUDA_CHECK(hipMalloc((void**)&gpuData->sumResult, sizeof(double)));

	// Image buffers
	CUDA_CHECK(hipMallocPitch(&gpuData->imgIn, &gpuData->pitchIn,
							gpuData->srcWidth*sizeof(uint8_t), gpuData->srcHeight));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgWeight, &gpuData->pitchWeight,
							gpuData->srcWidth*sizeof(uint8_t), gpuData->srcHeight));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgAccum, &gpuData->pitchAccum,
							gpuData->dstSize*sizeof(float), gpuData->dstSize));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgOut, &gpuData->pitchOutput,
							gpuData->dstSize*sizeof(uint8_t), gpuData->dstSize));

	printf("pitch imgIn:     %5lu (%lu) at %p\n", gpuData->pitchIn, 	gpuData->srcWidth*sizeof(uint8_t), gpuData->imgIn);
	printf("pitch imgWeight: %5lu (%lu) at %p\n", gpuData->pitchWeight, gpuData->srcWidth*sizeof(uint8_t), gpuData->imgWeight);
	printf("pitch Accum:     %5lu (%lu) at %p\n", gpuData->pitchAccum, 	gpuData->srcWidth*sizeof(float), gpuData->imgAccum);
	printf("pitch output:    %5lu (%lu) at %p\n", gpuData->pitchOutput, gpuData->srcWidth*sizeof(uint8_t), gpuData->imgOut);

	// Memory for 2D texture
	CUDA_CHECK(hipMallocPitch(&gpuData->lineCoverage, &gpuData->pitchCoverage,
							LINE_TEX_ANGLE_SAMPLES*sizeof(float), LINE_TEX_DIST_SAMPLES));

	printf("pitch lineCoverage: %5lu (%lu) at %p\n", gpuData->pitchCoverage, LINE_TEX_ANGLE_SAMPLES*sizeof(float), gpuData->lineCoverage);

	CUDA_CHECK(hipDeviceSynchronize());

	return CUDA_LAST_ERROR();
}

// Cleanup gpuImgData_t structure and free buffers in GPU memory
void GpuFreeBuffers(gpuData_t *gpuData) {
	printf("GpuFreeBuffers\n");

	CUDA_CHECK(hipStreamDestroy(gpuData->stream));
	CUDA_CHECK(hipDestroyTextureObject(gpuData->texCoverage));
	CUDA_CHECK(hipDestroyTextureObject(gpuData->texImageIn));
	CUDA_CHECK(hipDestroyTextureObject(gpuData->texWeights));

	// Free GPU memory
	if (gpuData->imgIn != NULL) 		CUDA_CHECK(hipFree(gpuData->imgIn));
	if (gpuData->imgWeight != NULL) 	CUDA_CHECK(hipFree(gpuData->imgWeight));
	if (gpuData->imgAccum != NULL) 		CUDA_CHECK(hipFree(gpuData->imgAccum));
	if (gpuData->imgOut != NULL) 		CUDA_CHECK(hipFree(gpuData->imgOut));

	if (gpuData->lineData != NULL) 		CUDA_CHECK(hipFree(gpuData->lineData));
	if (gpuData->lineCoverage != NULL) 	CUDA_CHECK(hipFree(gpuData->lineCoverage));

	if (gpuData->partialSums != NULL) 	CUDA_CHECK(hipFree(gpuData->partialSums));
	if (gpuData->sumResult != NULL) 	CUDA_CHECK(hipFree(gpuData->sumResult));
}

// Copy line data to GPU
void GpuLoadLines(gpuData_t *gpuData, line_t *lines) {

	CUDA_CHECK(hipMemcpy(gpuData->lineData, lines, NUM_LINES*4*sizeof(float), hipMemcpyHostToDevice));
	//CUDA_CHECK(hipDeviceSynchronize());
}

// Compute the perpendicular distance from the pixel (x0, y0) to the line Ax + By + C = 0
__device__ float compute_distance(float x0, float y0, float A, float B, float C, float inv_denom) {
	float numerator = fabsf(fmaf(A, x0, fmaf(B, y0, C)));  // |Ax0 + By0 + C| using fmaf for fused multiply-add
	return numerator * inv_denom;  // Multiply instead of divide
}

// Compute the angle of the line Ax + By + C = 0 relative to the x-axis
// Angle returned is in the range 0 to pi
__device__ float compute_angle(float A, float B) {
	// Calculate the angle in radians with respect to the x-axis
	float angle = atan2f(-A, B); // atan2(-A, B) ensures the correct quadrant

	// Convert the angle to the range [0, π] if necessary
	if (angle < 0) angle += (float)M_PI;

	return angle;
}

// (GPU) Draw many lines
__global__
void DrawLine_kernel(float *dataDst, size_t pitchDst, int width, int height, const float *lineData, float lineThickness, const hipTextureObject_t tex) {
	float A, B, C, inv_denom;
	float dist, angle;
	float maxDist;
	float value;
	int line;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// Flip vertically
	//int j_flip = height-1 - j;

	// Calculate the maximum distance at which the line overlaps a pixel
	maxDist = sqrtf(2)/2 + lineThickness/2;

	if ((i<width) && (j<height)) {
		value = 1.0f;

		for (line=0; line<NUM_LINES; line++) {
			// Get line parameters (format Ax + By + C = 0)
			// The parameter 1/sqrt(A^2 + B^2) is also precalculated
			A = lineData[4*line + 0];
			B = lineData[4*line + 1];
			C = lineData[4*line + 2];
			inv_denom = lineData[4*line + 3];

			// Calculate distance and angle for calculating partial coverage
			dist = compute_distance(i, j, A, B, C, inv_denom);
			angle = compute_angle(A, B);

			// Look up the coverage at this pixel and accumulate it to the total
			value *= (1.0f - tex2D<float>(tex, angle, dist/maxDist));
		}

		// Convert and store value into output array
		if (value < 1.0f) dataDst[j*pitchDst + i] = max(0.0f, value);
	}

}

// Draw many lines
void GpuDrawLines(gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	DrawLine_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgAccum, gpuData->pitchAccum/sizeof(float),
																width, height, gpuData->lineData, STRING_THICKNESS, gpuData->texCoverage);

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}


// Compute the sum of weighted errors for each block in the image
__global__ void computeBlockErrors_kernel(double* partialSums, const float* dataAccum, size_t pitchAccum, int width, int height,
									const hipTextureObject_t texImage, const hipTextureObject_t texWeight) {

	__shared__ double blockSum[SUM_BLOCK_SIZE*SUM_BLOCK_SIZE];

	float image, accum, weight;
	float diff;

	// Linear thread ID
	int tid = threadIdx.y * blockDim.x + threadIdx.x;

	// Pixel 2D position
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// Normalised 2D coordinates for textures
	float u = (float)i / (float)(width - 1);
	float v = (float)j / (float)(height - 1);

	// Fetch the data for this pixel
	image  = tex2D<float>(texImage, u, v);  // Input image
	weight = tex2D<float>(texWeight, u, v); // Weight
	accum  = dataAccum[j*pitchAccum + i];   // Generated line image

	// Compute the weighted absolute difference for the pixel
	diff = fabsf(image - accum)*weight;

	// Store the result in shared memory
	blockSum[tid] = diff;
	__syncthreads();

	// Perform reduction sum within the block
	for (int stride = (SUM_BLOCK_SIZE*SUM_BLOCK_SIZE) / 2; stride > 0; stride >>= 1) {
		if (tid < stride) {
			blockSum[tid] += blockSum[tid + stride];
		}
		__syncthreads();
	}

	// Write the block's sum to the global array
	if (tid == 0) {
		partialSums[blockIdx.y * gridDim.x + blockIdx.x] = blockSum[0];
	}

}

// Sum the block errors into a total error
__global__ void reducePartialSums_kernel(double* result, const double* partialSums, int numElements) {
	__shared__ double blockSum[SUM_BLOCK_SIZE*SUM_BLOCK_SIZE];

	int tx = threadIdx.x;
	int index = blockIdx.x * blockDim.x + tx;

	double sum = (index < numElements) ? partialSums[index] : 0.0;

	blockSum[tx] = sum;
	__syncthreads();

	// Perform reduction within the block
	for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
		if (tx < stride) {
			blockSum[tx] += blockSum[tx + stride];
		}
		__syncthreads();
	}

	// Write the block's sum to the global result
	if (tx == 0) {
		atomicAdd(result, blockSum[0]);
	}
}

// Compute the total weighted error between the original image and the generated lines
double GpucalculateImageError(gpuData_t *gpuData) {
	double h_result;

	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(SUM_BLOCK_SIZE, SUM_BLOCK_SIZE, 1);

	// Number of blocks (should alwyas be an integer multiple)
	const dim3 gridSize(width/blockSize.x, height/blockSize.y, 1);

	size_t numBlocks = gridSize.x * gridSize.y;

	// Clear the total sum in GPU memory
	hipMemset(gpuData->sumResult, 0, sizeof(double));
	CUDA_CHECK(hipDeviceSynchronize());//??

	// Launch the first kernel to compute block partial sums
	computeBlockErrors_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->partialSums, gpuData->imgAccum, gpuData->pitchAccum/sizeof(float), width, height,
									gpuData->texImageIn, gpuData->texWeights);

	//hipStreamSynchronize(gpuData->stream);  // Synchronize after first kernel??

	// Launch the second kernel to reduce partial sums
	int threadsPerBlock = 256;
	int blocksPerGrid = (numBlocks + threadsPerBlock - 1) / threadsPerBlock; // 16
	reducePartialSums_kernel<<<blocksPerGrid, threadsPerBlock, 0, gpuData->stream>>>(gpuData->sumResult, gpuData->partialSums, numBlocks);

	CUDA_CHECK(hipDeviceSynchronize());//??

	// Retrieve the final result
	hipMemcpyAsync(&h_result, gpuData->sumResult, sizeof(double), hipMemcpyDeviceToHost, gpuData->stream);

	return h_result;
}


// (GPU) Convert accumulator buffer to output format
__global__
void OutConvert_kernel(uint8_t *dataDst, size_t pitchDst, float *dataSrc, size_t pitchSrc, int width, int height) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// Flip vertically
	int j_flip = j;//height-1 - j;

	if ((i<width) && (j<height)) {
		// Convert and store value into output array
		dataDst[j*pitchDst + i] = round(dataSrc[j_flip*pitchSrc + i]*255.0f);
	}
}


// Set initial image stats
void GpuOutConvert(uint8_t *hostDst, gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	OutConvert_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgOut, gpuData->pitchOutput/sizeof(uint8_t),
																gpuData->imgAccum, gpuData->pitchAccum/sizeof(float), width, height);
	CUDA_CHECK(hipDeviceSynchronize());

	// Copy output image for display
	CUDA_CHECK(hipMemcpy2DAsync(
		hostDst, width * sizeof(uint8_t),
		gpuData->imgOut, gpuData->pitchOutput,
		width * sizeof(uint8_t), height, hipMemcpyDeviceToHost, gpuData->stream));

	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}

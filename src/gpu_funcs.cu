#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <stdint.h>
//#include <float.h>

// CUDA Runtime
#include <hip/hip_runtime.h>

#include "gpu_funcs.h"
#include "gpu_util.h"
#include "settings.h"

// GPU buffers
extern gpuData_t gpuData;

// Initialise gpuImgData_t structure and allocate buffers
int GpuInitBuffers(gpuData_t *gpuData) {
	printf("GpuInitBuffers\n");

	// Clear the structure in case of errors part way through initialisation
	memset(gpuData, 0, sizeof(gpuData_t));

	gpuData->srcWidth  = IMG_WIDTH;
	gpuData->srcHeight = IMG_HEIGHT;
	gpuData->dstSize = DATA_SIZE;

	// Create stream for processing
	CUDA_CHECK(hipStreamCreate(&gpuData->stream));

	CUDA_CHECK(hipMalloc(&gpuData->lineData, NUM_LINES*4*sizeof(float)));

	// Global memory on GPU
	CUDA_CHECK(hipMallocPitch(&gpuData->imgInOrig, &gpuData->pitchInOrig,
							gpuData->srcWidth*sizeof(uint8_t), gpuData->srcHeight));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgInFloat, &gpuData->pitchInFloat,
							gpuData->srcWidth*sizeof(float), gpuData->srcHeight));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgAccum, &gpuData->pitchAccum,
							gpuData->dstSize*sizeof(float), gpuData->dstSize));
	CUDA_CHECK(hipMallocPitch(&gpuData->imgOut, &gpuData->pitchOutput,
							gpuData->dstSize*sizeof(uint8_t), gpuData->dstSize));

	printf("pitch inOrig:  %5lu (%lu) at %p\n", gpuData->pitchInOrig, 	gpuData->srcWidth*sizeof(uint8_t), gpuData->imgInOrig);
	printf("pitch inFloat: %5lu (%lu) at %p\n", gpuData->pitchInFloat, 	gpuData->srcWidth*sizeof(float), gpuData->imgInFloat);
	printf("pitch Accum:   %5lu (%lu) at %p\n", gpuData->pitchAccum, 	gpuData->srcWidth*sizeof(float), gpuData->imgAccum);
	printf("pitch output:  %5lu (%lu) at %p\n", gpuData->pitchOutput, 	gpuData->srcWidth*sizeof(uint8_t), gpuData->imgOut);

	CUDA_CHECK(hipDeviceSynchronize());

	return CUDA_LAST_ERROR();
}

// Cleanup gpuImgData_t structure and free buffers in GPU memory
void GpuFreeBuffers(gpuData_t *gpuData) {
	printf("GpuFreeBuffers\n");

	CUDA_CHECK(hipStreamDestroy(gpuData->stream));

	// Free GPU memory
	if (gpuData->lineData != NULL) 		CUDA_CHECK(hipFree(gpuData->lineData));
	if (gpuData->imgInOrig != NULL) 	CUDA_CHECK(hipFree(gpuData->imgInOrig));
	if (gpuData->imgInFloat != NULL) 	CUDA_CHECK(hipFree(gpuData->imgInFloat));
	if (gpuData->imgAccum != NULL) 		CUDA_CHECK(hipFree(gpuData->imgAccum));
	if (gpuData->imgOut != NULL) 		CUDA_CHECK(hipFree(gpuData->imgOut));
}


// Copy line data to GPU
void GpuLoadLines(gpuData_t *gpuData, line_t *lines) {

	CUDA_CHECK(hipMemcpy(gpuData->lineData, lines, NUM_LINES*4*sizeof(float), hipMemcpyHostToDevice));
	//CUDA_CHECK(hipDeviceSynchronize());
}

// Compute the perpendicular distance from the pixel (x0, y0) to the line Ax + By + C = 0
__device__ float compute_distance(float x0, float y0, float A, float B, float C, float inv_denom) {
	float numerator = fabsf(fmaf(A, x0, fmaf(B, y0, C)));  // |Ax0 + By0 + C| using fmaf for fused multiply-add
	return numerator * inv_denom;  // Multiply instead of divide
}

// (GPU) Draw a line
__global__
void DrawLine_kernel(float *dataDst, size_t pitchDst, int width, int height, const float *lineData) {
	float A, B, C, inv_denom;
	float x, y, dist;
	float value;

	// Get line parameters (format Ax + By + C = 0)
	// The parameter 1/sqrt(A^2 + B^2) is also precalculated
	A = lineData[0];
	B = lineData[1];
	C = lineData[2];
	inv_denom = lineData[3];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	x = i;
	y = j;

	if ((i<width) && (j<height)) {
		dist = compute_distance(x, y, A, B, C, inv_denom);

		//if (dist < 1.0f) value = 0.7f;
		//else value = 0.2f;
		value = min(1.0f, max(0.0f, 1.5f - dist/2));

		// Convert and store value into output array
		if (value > 0.0f) dataDst[j*pitchDst + i] = value;
	}
}



// Draw a line
void GpuDrawLines(gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	DrawLine_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgAccum, gpuData->pitchAccum/sizeof(float),
																width, height, gpuData->lineData);

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}





// (GPU) Convert accumulator buffer to output format
__global__
void OutConvert_kernel(uint8_t *dataDst, size_t pitchDst, float *dataSrc, size_t pitchSrc, int width, int height) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// Flip vertically
	int j_flip = height-1 - j;

	if ((i<width) && (j<height)) {
		// Convert and store value into output array
		dataDst[j*pitchDst + i] = round(dataSrc[j_flip*pitchSrc + i]*255.0f);
	}
}

// Set initial image stats
void GpuOutConvert(uint8_t *hostDst, gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	OutConvert_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgOut, gpuData->pitchOutput/sizeof(uint8_t),
																gpuData->imgAccum, gpuData->pitchAccum/sizeof(float), width, height);
	CUDA_CHECK(hipDeviceSynchronize());

	// Copy output image for display
	CUDA_CHECK(hipMemcpy2DAsync(
		hostDst, width * sizeof(uint8_t),
		gpuData->imgOut, gpuData->pitchOutput,
		width * sizeof(uint8_t), height, hipMemcpyDeviceToHost, gpuData->stream));

	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}

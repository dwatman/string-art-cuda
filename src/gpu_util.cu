#include "hip/hip_runtime.h"
#include <stdio.h>

#include "gpu_util.h"
#include "settings.h"

// CPU memory buffers
extern uint8_t *h_imageIn;
extern uint8_t *h_imageOut;
extern float   *h_lineCoverage;

// GPU memory buffers
extern gpuData_t gpuData;

// Check enclosed function for CUDA errors
hipError_t cudaCheck(hipError_t err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		fprintf(stderr,"ERROR in CUDA Runtime at: %s line %d\n", file, line);
		fprintf(stderr,"    (%d) %s\n", err, hipGetErrorString(err));
		// Don't exit on error
		// exit(EXIT_FAILURE);
	}
	return err;
}

// Check last recorded CUDA status for errors and clear non-sticky error flag
hipError_t cudaLastError(const char* const file, const int line) {
	hipError_t const err{hipGetLastError()};

	if (err != hipSuccess) {
		fprintf(stderr,"ERROR in CUDA Runtime at: %s line %d\n", file, line);
		fprintf(stderr,"    (%d) %s\n", err, hipGetErrorString(err));
		// Don't exit on error
		// exit(EXIT_FAILURE);
	}
	return err;
}

// Initialise GPU and buffers
int GpuInit(void) {
	int err;
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;
	int deviceCount = 0;
	int driverVersion = 0, runtimeVersion = 0;

	// Check for available GPUs (returns 0 if there are no CUDA capable devices)
	cudaStatus = CUDA_CHECK(hipGetDeviceCount(&deviceCount));
	if (cudaStatus != hipSuccess) {
		exit(EXIT_FAILURE);
	}

	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
		exit(EXIT_FAILURE);
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	// Use first device
	CUDA_CHECK(hipSetDevice(0));
	CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
	CUDA_CHECK(hipDriverGetVersion(&driverVersion));
	CUDA_CHECK(hipRuntimeGetVersion(&runtimeVersion));

	printf("Device %d: \"%s\"\n", 0, deviceProp.name);
	printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
		driverVersion / 1000, (driverVersion % 100) / 10,
		runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
		deviceProp.major, deviceProp.minor);

	printf("texturePitchAlignment: %lu\n", deviceProp.texturePitchAlignment);

	err = CUDA_LAST_ERROR();
	printf("CUDA initialized\n\n");

	return err;
}

// Page-lock/pin CPU memory for faster access by GPU
void GpuPinMemory(void *ptr, size_t size) {
	CUDA_CHECK(hipHostRegister(ptr, size, 0));
}

// Unpin CPU memory before freeing
void GpuUnPinMemory(void *ptr) {
	CUDA_CHECK(hipHostUnregister(ptr));
}

// Allocate aligned memory on CPU and pin it for fast GPU access
int AllocAndAlignPinned(void **buf, size_t size) {
	size_t sizeAligned;

	// Use fixed 4096 byte alignment to match page size, as GPU pins whole pages (probably)
	size_t alignment = 4096;

	// Make sure the buffer is a multiple of the alignment size
	sizeAligned  = (((alignment-1) + size) / alignment) * alignment;

	//printf("size:  %lu (%lu aligned)\n", size, sizeAligned);

	// Allocate aligned memory on CPU
	*buf = aligned_alloc(alignment, sizeAligned);

	if (*buf == NULL) {
		printf("Error in AllocAndAlignPinned, could not allocate aligned buffer\n");
		return -1;
	}

	// Pin aligned memory for faster GPU access
	GpuPinMemory(*buf, sizeAligned);

	return CUDA_LAST_ERROR();
}

// Initialise pinned host memory
int InitPinnedBuffers(void) {
	printf("InitPinnedBuffers\n");

	AllocAndAlignPinned((void **)&h_imageIn, IMG_WIDTH * IMG_HEIGHT * sizeof(uint8_t));
	AllocAndAlignPinned((void **)&h_imageOut, DATA_SIZE * DATA_SIZE * sizeof(uint8_t));
	AllocAndAlignPinned((void **)&h_lineCoverage, LINE_TEX_ANGLE_SAMPLES * LINE_TEX_DIST_SAMPLES * sizeof(float));

	printf("h_imageIn at      %p\n", h_imageIn);
	printf("h_imageOut at     %p\n", h_imageOut);
	printf("h_lineCoverage at %p\n", h_lineCoverage);

	return CUDA_LAST_ERROR();
}

// Free pinned host memory
void FreePinnedBuffers(void) {
	printf("FreePinnedBuffers\n");

	if (h_imageIn != NULL) {
		GpuUnPinMemory(h_imageIn);
		free(h_imageIn);
	}

	if (h_imageOut != NULL) {
		GpuUnPinMemory(h_imageOut);
		free(h_imageOut);
	}

	if (h_lineCoverage != NULL) {
		GpuUnPinMemory(h_lineCoverage);
		free(h_lineCoverage);
	}
}

// Wait for GPU to finish before accessing on host
void GpuSync(void) {
	hipDeviceSynchronize();

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}

// (CUDA) Fill buffer with a value
__global__
void fill_kernel(float *dataDst, size_t pitchDst, float value, int width, int height) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i<width) && (j<height)) {
		// Store value into output array
		dataDst[j*pitchDst + i] = value;
	}
}

// Set initial image stats
void ClearBuffers(gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	// Set accumulator to black
	fill_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgAccum, gpuData->pitchAccum/sizeof(float), 0.0f, width, height);

	//CUDA_CHECK(hipMemset2DAsync(gpuData->imgOut, gpuData->pitchOutput, 0x10, width, height, gpuData->stream));
	CUDA_CHECK(hipDeviceSynchronize());
}

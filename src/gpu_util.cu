#include "hip/hip_runtime.h"
#include <stdio.h>

#include "gpu_util.h"
#include "settings.h"

// CPU memory buffers
extern uint8_t *h_imageIn;
extern uint8_t *h_weights;
extern uint8_t *h_imageOut;
extern float   *h_lineCoverage;

// GPU memory buffers
extern gpuData_t gpuData;

// Check enclosed function for CUDA errors
hipError_t cudaCheck(hipError_t err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		fprintf(stderr,"ERROR in CUDA Runtime at: %s line %d\n", file, line);
		fprintf(stderr,"    (%d) %s\n", err, hipGetErrorString(err));
		// Don't exit on error
		// exit(EXIT_FAILURE);
	}
	return err;
}

// Check last recorded CUDA status for errors and clear non-sticky error flag
hipError_t cudaLastError(const char* const file, const int line) {
	hipError_t const err{hipGetLastError()};

	if (err != hipSuccess) {
		fprintf(stderr,"ERROR in CUDA Runtime at: %s line %d\n", file, line);
		fprintf(stderr,"    (%d) %s\n", err, hipGetErrorString(err));
		// Don't exit on error
		// exit(EXIT_FAILURE);
	}
	return err;
}

// Initialise GPU and buffers
int GpuInit(void) {
	int err;
	hipError_t cudaStatus;
	hipDeviceProp_t deviceProp;
	int deviceCount = 0;
	int driverVersion = 0, runtimeVersion = 0;

	// Check for available GPUs (returns 0 if there are no CUDA capable devices)
	cudaStatus = CUDA_CHECK(hipGetDeviceCount(&deviceCount));
	if (cudaStatus != hipSuccess) {
		exit(EXIT_FAILURE);
	}

	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
		exit(EXIT_FAILURE);
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	// Use first device
	CUDA_CHECK(hipSetDevice(0));
	CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
	CUDA_CHECK(hipDriverGetVersion(&driverVersion));
	CUDA_CHECK(hipRuntimeGetVersion(&runtimeVersion));

	printf("Device %d: \"%s\"\n", 0, deviceProp.name);
	printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
		driverVersion / 1000, (driverVersion % 100) / 10,
		runtimeVersion / 1000, (runtimeVersion % 100) / 10);
	printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
		deviceProp.major, deviceProp.minor);

	printf("texturePitchAlignment: %lu\n", deviceProp.texturePitchAlignment);

	err = CUDA_LAST_ERROR();
	printf("CUDA initialized\n\n");

	return err;
}

// Page-lock/pin CPU memory for faster access by GPU
void GpuPinMemory(void *ptr, size_t size) {
	CUDA_CHECK(hipHostRegister(ptr, size, 0));
}

// Unpin CPU memory before freeing
void GpuUnPinMemory(void *ptr) {
	CUDA_CHECK(hipHostUnregister(ptr));
}

// Allocate aligned memory on CPU and pin it for fast GPU access
int AllocAndAlignPinned(void **buf, size_t size) {
	size_t sizeAligned;

	// Use fixed 4096 byte alignment to match page size, as GPU pins whole pages (probably)
	size_t alignment = 4096;

	// Make sure the buffer is a multiple of the alignment size
	sizeAligned  = (((alignment-1) + size) / alignment) * alignment;

	//printf("size:  %lu (%lu aligned)\n", size, sizeAligned);

	// Allocate aligned memory on CPU
	*buf = aligned_alloc(alignment, sizeAligned);

	if (*buf == NULL) {
		printf("Error in AllocAndAlignPinned, could not allocate aligned buffer\n");
		return -1;
	}

	// Pin aligned memory for faster GPU access
	GpuPinMemory(*buf, sizeAligned);

	return CUDA_LAST_ERROR();
}

// Initialise pinned host memory
int InitPinnedBuffers(gpuData_t *gpuData) {
	printf("InitPinnedBuffers\n");

	// imageIn is already allocated so pin it only
	GpuPinMemory(h_imageIn, gpuData->srcWidth * gpuData->srcHeight * sizeof(uint8_t));
	//AllocAndAlignPinned((void **)&h_imageIn, gpuData->srcWidth * gpuData->srcHeight * sizeof(uint8_t));

	AllocAndAlignPinned((void **)&h_imageOut, DATA_SIZE * DATA_SIZE * sizeof(uint8_t));
	AllocAndAlignPinned((void **)&h_lineCoverage, LINE_TEX_DIST_SAMPLES * LINE_TEX_ANGLE_SAMPLES * sizeof(float));

	printf("h_imageIn at      %p\n", h_imageIn);
	printf("h_imageOut at     %p\n", h_imageOut);
	printf("h_lineCoverage at %p\n", h_lineCoverage);

	return CUDA_LAST_ERROR();
}

// Free pinned host memory
void FreePinnedBuffers(void) {
	printf("FreePinnedBuffers\n");

	if (h_imageIn != NULL) {
		GpuUnPinMemory(h_imageIn);
		free(h_imageIn);
	}

	if (h_weights != NULL) {
		//GpuUnPinMemory(h_weights);
		free(h_weights);
	}

	if (h_imageOut != NULL) {
		GpuUnPinMemory(h_imageOut);
		free(h_imageOut);
	}

	if (h_lineCoverage != NULL) {
		GpuUnPinMemory(h_lineCoverage);
		free(h_lineCoverage);
	}
}

// Create a bindless texture for the input image
void InitImageInTexture(gpuData_t *deviceData) {
	hipResourceDesc texRes;
	hipTextureDesc texDescr;

	// Clear resource descriptors
	memset(&texRes, 0, sizeof(hipResourceDesc));
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	// Set up the 2D texture parameters
	texRes.resType = hipResourceTypePitch2D;
	texRes.res.pitch2D.devPtr = (void *)deviceData->imgIn;
	texRes.res.pitch2D.desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	texRes.res.pitch2D.width = deviceData->srcWidth;
	texRes.res.pitch2D.height = deviceData->srcHeight;
	texRes.res.pitch2D.pitchInBytes = deviceData->pitchIn;

	// Set up the way the texture is accessed
	texDescr.normalizedCoords = 1;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeNormalizedFloat;

	CUDA_CHECK(hipCreateTextureObject(&deviceData->texImageIn, &texRes, &texDescr, NULL));
}

// Create a bindless texture for the image weights
void InitWeightsTexture(gpuData_t *deviceData) {
	hipResourceDesc texRes;
	hipTextureDesc texDescr;

	// Clear resource descriptors
	memset(&texRes, 0, sizeof(hipResourceDesc));
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	// Set up the 2D texture parameters
	texRes.resType = hipResourceTypePitch2D;
	texRes.res.pitch2D.devPtr = (void *)deviceData->imgWeight;
	texRes.res.pitch2D.desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	texRes.res.pitch2D.width = deviceData->srcWidth;
	texRes.res.pitch2D.height = deviceData->srcHeight;
	texRes.res.pitch2D.pitchInBytes = deviceData->pitchWeight;

	// Set up the way the texture is accessed
	texDescr.normalizedCoords = 1;
	texDescr.filterMode = hipFilterModeLinear;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeNormalizedFloat;

	CUDA_CHECK(hipCreateTextureObject(&deviceData->texWeights, &texRes, &texDescr, NULL));
}

// Copy line coverage data to GPU
void GpuUpdateCoverage(gpuData_t *deviceData, const float *hostData) {
	CUDA_CHECK(hipMemcpy2DAsync(
		deviceData->lineCoverage, deviceData->pitchCoverage,
		hostData, LINE_TEX_DIST_SAMPLES*sizeof(float),
		LINE_TEX_DIST_SAMPLES*sizeof(float), LINE_TEX_ANGLE_SAMPLES,
		hipMemcpyHostToDevice, deviceData->stream));
}

// Copy input image to GPU
void GpuUpdateImageIn(gpuData_t *deviceData, const uint8_t *hostData) {
	CUDA_CHECK(hipMemcpy2DAsync(
		deviceData->imgIn, deviceData->pitchIn,
		hostData, deviceData->srcWidth*sizeof(uint8_t),
		deviceData->srcWidth*sizeof(uint8_t), deviceData->srcHeight,
		hipMemcpyHostToDevice, deviceData->stream));
}

// Copy weighting mask to GPU
void GpuUpdateWeights(gpuData_t *deviceData, const uint8_t *hostData) {
	CUDA_CHECK(hipMemcpy2DAsync(
		deviceData->imgWeight, deviceData->pitchWeight,
		hostData, deviceData->srcWidth*sizeof(uint8_t),
		deviceData->srcWidth*sizeof(uint8_t), deviceData->srcHeight,
		hipMemcpyHostToDevice, deviceData->stream));
}

// Wait for GPU to finish before accessing on host
void GpuSync(void) {
	hipDeviceSynchronize();

	CUDA_LAST_ERROR(); // Clear previous non-sticky errors
}

// (CUDA) Fill buffer with a value
__global__
void fill_kernel(float *dataDst, size_t pitchDst, float value, int width, int height) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i<width) && (j<height)) {
		// Store value into output array
		dataDst[j*pitchDst + i] = value;
	}
}

// Clear the accumulator buffer
void ClearAccumBuffer(gpuData_t *gpuData) {
	int width = gpuData->dstSize;
	int height = gpuData->dstSize;

	// Threads per Block
	const dim3 blockSize(16,16,1);

	// Number of blocks
	const dim3 gridSize(ceil(width/(float)blockSize.x),
						ceil(height/(float)blockSize.y),
						1);

	// Set accumulator to white
	fill_kernel<<<gridSize, blockSize, 0, gpuData->stream>>>(gpuData->imgAccum, gpuData->pitchAccum/sizeof(float), 1.0f, width, height);

	CUDA_CHECK(hipDeviceSynchronize());
}
